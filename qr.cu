#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

//Overall algorithm:
//For each panel column (SERIAL):
//  For each dense panel from the bottom up (SERIAL):
//    Load panel into shared
//    For each panel column (SERIAL):
//      compute and store hh reflector and beta
//      apply reflectors to trailing columns of panel (possibly parallel)
//      reflectors are written back to lower part of panel,
//    For each trailing panel (PARALLEL):
//      apply the reflections to trailing panel, replacing values

//scalar type and panel size (RxC)
#define Scalar float
#define R 128
#define C 32

extern __shared__ float currentPanel[R][C];

//(2RC + C) * sizeof(Scalar) must fit in 48 KiB
//mat should be column-major
__global__ void mmqr(Scalar* mat, int m, int n)
{
  //iterate over all subdiagonal panels
  //first left to right
  for(int pc = 0; pc < n; pc += C)
  {
    //then bottom to top, sliding panel up by R-C each iteration
    //TODO: in between iterations, keep the overlapping rows in shared mem
    for(int pr = m - R; pr >= pc; pr -= (R-C))
    {
      //load panel into shared

    }
  }
}

//left and right are RxC arrays
//left is the current panel being eliminated
//right is a panel in the trailing matrix
__device__ void applyPanel()
{
}

__global__ void add(int* a, int* b, int* c)
{
  int i = blockIdx.x;
  c[i] = a[i] + b[i];
}

int main()
{
  int* a;
  int* b;
  int* c;
  hipMalloc((void**) &a, R);
  hipMalloc((void**) &b, R);
  hipMalloc((void**) &c, R);
  add<<<R, 1>>>(a, b, c);
  hipFree(a);
  hipFree(b);
  hipFree(c);
  return 0;
}

